#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <chrono>
#include <string>
#include <sstream>
#include <iostream>
#include <vector>
#include <fstream>
#include "parser.hpp"

using namespace std;
using namespace aria::csv;


hipError_t decomposeCuda(float *A, int size, int threads_per_block, int num_blocks);


float * read_csv(string path, int N) {
	/**
	 * Reads a csv files and creates a matrix
	*/

	std::ifstream f(path);
	CsvParser parser = CsvParser(f).delimiter(',');

	int i = 0, j = 0;
	float *A = (float *)malloc(sizeof(float) * N * N);
	for (auto& row : parser) {
		j = 0;
		for (auto& field : row) {
			A[i * N + j] = std::stof(field);
			j++;
		}
		i++;
	}

	return A;
}

float **matrix_create(size_t m, size_t n) {
	float **result = (float **)malloc(sizeof(float *)*m);

	for (int i = 0; i < m; i++) {
		result[i] = (float *)malloc(sizeof(float)*n);
	}
	return result;
}

float **matrix_difference(float **ma, float **mb, int n) {

	float **mc = matrix_create(n, n);

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			mc[i][j] = ma[i][j] - mb[i][j];
		}
	}

	return mc;
}

float **matrix_multiplication(float **ma, float **mb, int n) {
	float **mc = matrix_create(n, n);

	float sum = 0;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			sum = 0;
			for (int k = 0; k < n; k++) {
				sum = sum + ma[i][k] * mb[k][j];
			}
			mc[i][j] = sum;
		}
	}
	return mc;
}


double frobenius_norm(float **A, int n) {
	float sum = 0;

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			sum += A[i][j] * A[i][j];
		}
	}

	double norm = sqrt(sum);
	return norm;
}


typedef struct {
	float **L;
	float **U;
} LU;


float **initialize_matrix(int rows, int cols) {

	float **matrix = matrix_create(rows, cols);

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			matrix[i][j] = 0;
		}
	}
	return matrix;
}

LU split_lu(float **a, int n) {
	float **L = initialize_matrix(n, n);
	float **U = initialize_matrix(n, n);


	int i, j;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			if (i < j)
			{
				U[i][j] = a[i][j];
			}
			if (i > j) {
				L[i][j] = a[i][j];
			}
			if (i == j) {
				L[i][j] = 1;
				U[i][j] = a[i][j];
			}
		}
	}


	LU decomposition = LU();
	decomposition.L = L;
	decomposition.U = U;
	return decomposition;
}

double compute_error(float **a, float **b, int n) {
	float **difference = matrix_difference(a, b, n);
	double error = frobenius_norm(difference, n);
	return error;
}



__constant__ int MATRIX_SIZE;


__global__ void decompose_multipliers(float *A, int rows_per_thread, int i) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int jstart = (i + 1) + tid * rows_per_thread;
	int jend = jstart + rows_per_thread;

	for (int j = jstart; j < jend && j < MATRIX_SIZE; j++) {
		A[j * MATRIX_SIZE + i] = A[j * MATRIX_SIZE + i] / A[i * MATRIX_SIZE + i]; // Computes the multipliers and updates L in A
	}
}

__global__ void decompose_elimination(float *A, int rows_per_thread, int i) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int jstart = (i + 1) + tid * rows_per_thread;
	int jend = jstart + rows_per_thread;

	for (int j = jstart; j < jend && j < MATRIX_SIZE; j++) { // Iterates over the remaining rows
		for (int k = i + 1; k < MATRIX_SIZE; k++) { // iterates over the remaining columns
			A[j * MATRIX_SIZE + k] -= A[j * MATRIX_SIZE + i] * A[i * MATRIX_SIZE + k]; // Updates U in A
		}
	}

}

void print_matrix(float **matrix, int n) {
	/**
	 * Prints the matrix
	 * @param matrix The matrix to print
	*/


	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf("%f\t", matrix[i][j]);
		}
		printf("\n");
	}
}

float **matrix2d(float *A, int n) {
	float **original = matrix_create(n, n);
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			original[i][j] = A[i*n + j];
		}
	}
	return original;
}



int main(int argc, char const *argv[])
{
	int size = atoi(argv[1]);
	//int size = 2000;
	char path[255];
	int out = snprintf(path, 255, "C:\\Users\\sasce\\Desktop\\Matrices\\matrix_%i.csv", size);

	float *A = read_csv(path, size);
	float **original = matrix2d(A, size);

	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	// cout << "using " << properties.multiProcessorCount << " multiprocessors" << endl;
	// cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << endl;

	// Decomplse matrix in parallel.
	int threads_per_block = atoi(argv[2]);
	int num_blocks = atoi(argv[3]);
	// int threads_per_block = 512;
	// int num_blocks = 2;
	auto start = std::chrono::high_resolution_clock::now();
	hipError_t cudaStatus = decomposeCuda(A, size, threads_per_block, num_blocks);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "decomposeCuda failed!");
		return 1;
	}

	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = finish - start;

	cout << size << ";" << num_blocks << ";" << threads_per_block << ";" << (int)elapsed.count() << "\n";

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}


	/*
	float **result = matrix2d(A, size);
	LU decomposition = split_lu(result, size);
	float **recomposed = matrix_multiplication(decomposition.L, decomposition.U, size);
	double error = compute_error(recomposed, original, size);
	printf("Error %f", error);
	*/
	return 0;
}

// Helper function for using CUDA to decompose matrix in parallel.
hipError_t decomposeCuda(float *A, int size, int threads_per_block, int num_blocks)
{
	float *dev_a;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_a, size * size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, A, size * size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	// Copy input size from host memory to GPU.
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(MATRIX_SIZE), &size, sizeof(size));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.

	int ops_per_thread = ceil((size) / (float)(threads_per_block*num_blocks));

	dim3 thread_block(threads_per_block, 1, 1);
	dim3 grid(num_blocks, 1);
	// printf("Ops per thread %i \n", ops_per_thread);

	for (int i = 0; i < size; i++) { // Iterates over the columns to remove
		decompose_multipliers << <grid, thread_block >> > (dev_a, ops_per_thread, i);
		decompose_elimination << <grid, thread_block >> > (dev_a, ops_per_thread, i);
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "decomposeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching decomposeKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(A, dev_a, size * size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);

	return cudaStatus;
}